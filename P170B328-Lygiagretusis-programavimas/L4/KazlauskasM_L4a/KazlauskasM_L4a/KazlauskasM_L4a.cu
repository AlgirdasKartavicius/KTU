#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>

using namespace std;

/* Konstantos */
/* Masyv� skai�ius*/
const int N = 6;
/* Automobili� (�ra��) skai�ius masyve */
const int K = 12;
/* Maksimalus automobilio pavadinimo ilgis */
const int PAVADINIMO_ILGIS_MAX = 15;

/* Automobilio strukt�ra */
struct Automobilis{
	public:
		// N pavadinim� po PAVADINIMO_ILGIS_MAX raid�i� ir dar +1 eilut�s pabaigos simboliui
		char pavadinimas[N * PAVADINIMO_ILGIS_MAX + 1];
		int metai;
		double litrai;

		/* Konstruktorius, priskiriantis strukt�ros kintamiesiems tu��ias reik�mes
		__host__ - konstruktorius gali b�ti kvie�iamas i� CPU */
		__host__ Automobilis() : metai(0), litrai(0.0){ 
			memset(pavadinimas, ' ', N * PAVADINIMO_ILGIS_MAX - 1); pavadinimas[N * PAVADINIMO_ILGIS_MAX] = '\0';
		};

		/* Destruktorius 
		__host__ ir __device__ - destruktorius gali b�ti kvie�iamas tiek i� CPU, tiek i� GPU*/
		__host__ __device__ ~Automobilis() {};

		/* Konstruktorius su parametrais 
		__device__ - konstruktorius gali b�ti kvie�iamas i� GPU*/
		__device__ Automobilis(char pavadinimas[], int metai, double litrai){
			for (int i = 0; i < N * PAVADINIMO_ILGIS_MAX; i++) this->pavadinimas[i] = pavadinimas[i];
			this->metai = metai;
			this->litrai = litrai;
		}	
};
/* Programos vykdymo funkcija, kurioje i�skiriama atmintis GPU, ten atliekami veiksmai, gr��inami rezultatai
	gr��ina funcijos status�, t.y., hipSuccess, jei viskas pavyko gerai */
hipError_t vykdyti(Automobilis **duomenys, Automobilis *rezultatai);
/* Funkcija, sudedanti atitinkam� masyv� element� lauk� reik�mes */
__global__ void sudeti(Automobilis *automobiliai, Automobilis *rezultatai);
/* Duomen� skaitymo funkcija */
void skaityti(Automobilis** automobiliai);
/* Pradini� duomen� spausdinimo funkcija */
void spausdintiDuomenis(Automobilis** automobiliai);
/* Rezultat� spausdinimo funkcija */
void spausdintiRezultatus(Automobilis *automobiliai);

int main()
{
	/* Automobili� duomen� dvimatis masyvas*/
	Automobilis** automobiliai = new Automobilis*[N];
	skaityti(automobiliai);
	
	/* Rezultat� masyvas */
	Automobilis *rezultatai = new Automobilis[K];

	hipError_t cudaStatus = vykdyti(automobiliai, rezultatai);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vykdyti failed!");
        return 1;
    }

	/* � rezultat� fail� pausdinami pradiniai duomenys bei rezultatai*/
	spausdintiDuomenis(automobiliai);
	spausdintiRezultatus(rezultatai);

	/* Kodo dalis, reikalinga Nsight ir Visual Profiler �rankiams*/
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	/* Atlaisvinama dinami�kai i�skirta atmintis */
	delete[] automobiliai;
	delete[] rezultatai;
    return 0;
}

hipError_t vykdyti(Automobilis **duomenys_matrica, Automobilis *rezultatai)
{
	/* CUDA statusas*/
	hipError_t cudaStatus;

	/* GPU skirti kintamieji*/
	Automobilis *dev_rezultatai = new Automobilis[K];
	Automobilis *dev_duomenys = new Automobilis[K * N];

	/* Duomen� matrica i�skleid�iama � duomen� masyv� (vien� eilut�)*/
	Automobilis *duomenys_masyvas = new Automobilis[K * N];
	for (int i = 0; i < N; i++)
		for (int j = 0; j < K; j++)
			duomenys_masyvas[i * K + j] = duomenys_matrica[i][j];

    // Pasirenkama, kuriame GPU �renginyje leisti program�
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // I�skiriama atmintis GPU �renginyje   .
    cudaStatus = hipMalloc((void**)&dev_duomenys, N * K * sizeof(Automobilis));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rezultatai, K * sizeof(Automobilis));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Nukopijuojami duomenys � GPU kintamuosius
    cudaStatus = hipMemcpy(dev_duomenys, duomenys_masyvas, N * K * sizeof(Automobilis), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rezultatai, rezultatai, K * sizeof(Automobilis), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Vykdoma programa lygiagre�iai 1 bloke, gij� skai�ius - K
	sudeti<< <1, K>> >(dev_duomenys, dev_rezultatai);

    // Tikrinama, ar lygiagretaus kodo vykdymo metu atsirado koki� klaid�
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize laukiama funkcijos vykdymo pabaigos
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Kopijuojami rezultatai i� GPU � CPU
    cudaStatus = hipMemcpy(rezultatai, dev_rezultatai, K * sizeof(Automobilis), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
	/* Atlaisvinama atmintis */
	delete[] duomenys_masyvas;
    hipFree(dev_duomenys);
    hipFree(dev_rezultatai);
    
    return cudaStatus;
}

__global__ void sudeti(Automobilis *automobiliai, Automobilis *rezultatai)
{
	/* Paimamas gijos indeksas (kadangi masyvas - vienmatis, imama x koordinat�)*/
	int id = threadIdx.x;
	int metai = 0;
	double litrai = 0.0;
	char pavadinimai[N * PAVADINIMO_ILGIS_MAX];
	/* Einama per vis� masyv� */
	for (int i = 0; i < N; i++){
		/* Kadangi matrica buvo i�reik�ta eilute, tai jei matricoje �ra�o koordinat�s buvo
			automobiliai[i][j], kur i - eilut�, j - stulpelis, masyve �ias koordinates atitinka
			automobiliai[i * K + j], kur K - �ra�� kiekis viename masyve arba �ra�� kiekis
			buvusioje matricos eilut�je */
		metai += automobiliai[i * K + id].metai;
		litrai += automobiliai[i * K + id].litrai;
		for (int j = 0; j < PAVADINIMO_ILGIS_MAX; j++) pavadinimai[PAVADINIMO_ILGIS_MAX * i + j] = automobiliai[i * K + id].pavadinimas[j];
	}
	rezultatai[id] = Automobilis(pavadinimai, metai, litrai);
}

void skaityti(Automobilis** automobiliai){
	ifstream F("KazlauskasM_L4.txt");
	string pavadinimas;
	for (int i = 0; i < N; i++){
		Automobilis *automobiliai_temp = new Automobilis[K];
		F.ignore();
		for (int j = 0; j < K; j++){
			F >> pavadinimas;
			for (unsigned int k = 0; k < pavadinimas.length(); k++) automobiliai_temp[j].pavadinimas[k] = pavadinimas[k];
			F >> automobiliai_temp[j].metai >> automobiliai_temp[j].litrai;
			F.ignore();
		}
		automobiliai[i] = automobiliai_temp;
	}
	F.close();
}
void spausdintiDuomenis(Automobilis** automobiliai){
	ofstream R("KazlauskasM_L4a_rez.txt");
	int masyvo_nr = 1;
	for (int i = 0; i < N; i++){
		int lineNr = 1;
		R << "****** Automobili� masyvas Nr. " << masyvo_nr++ << " ******" << endl;
		R << "   |" << string(PAVADINIMO_ILGIS_MAX, '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
		R << "   |" << setw(PAVADINIMO_ILGIS_MAX) << left << "Pavadinimas" << setw(13) << left << "|Metai" << setw(9) << left << "|Litrai   |" << endl;
		R << "   |" << string(PAVADINIMO_ILGIS_MAX, '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
		for (int j = 0; j < K; j++){
			R << setw(3) << left << lineNr++ << "|";
			for (int k = 0; k < PAVADINIMO_ILGIS_MAX; k++) R << automobiliai[i][j].pavadinimas[k];
			R << "|" << setw(12) << left << automobiliai[i][j].metai << "|";
			R << setw(9) << left << fixed << setprecision(2) << automobiliai[i][j].litrai << "|" << endl;
		}
		R << "   |" << string(PAVADINIMO_ILGIS_MAX, '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
		R << endl;
	}
}
void spausdintiRezultatus(Automobilis *automobiliai){
	ofstream R("KazlauskasM_L4a_rez.txt", ios::app);
	int lineNr = 1;
	R << "*******************************************" << endl;
	R << "Rezultatai" << endl;
	R << "*******************************************" << endl;
	R << "   |" << string((N * PAVADINIMO_ILGIS_MAX), '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
	R << "   |" << setw(N * PAVADINIMO_ILGIS_MAX) << left << "Sujungti pavadinimai" << setw(13) << left << "|Metai" << setw(9) << left << "|Litrai   |" << endl;
	R << "   |" << string((N * PAVADINIMO_ILGIS_MAX), '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
	for (int i = 0; i < K; i++){
		R << setw(3) << left << lineNr++ << "|" << setw(N * PAVADINIMO_ILGIS_MAX) << left << automobiliai[i].pavadinimas;
		R << "|" << setw(12) << left << automobiliai[i].metai << "|";
		R << setw(9) << left << fixed << setprecision(2) << automobiliai[i].litrai << "|" << endl;
	}
	R << "   |" << string((N * PAVADINIMO_ILGIS_MAX), '-') << "|" << string(12, '-') << "|" << string(9, '-') << "|" << endl;
	R.close();
}
