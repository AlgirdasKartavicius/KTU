#include "hip/hip_runtime.h"
/*
Laboratorinio darbo u�duotis: L1c
Mangirdas Kazlauskas IFF-4/1
Atsakymai � klausimus:
1) Visi vienu metu (tokio atsakymo varianto n�ra, bet pagal teorij� yra taip)
2) Atsitiktine
3) Vien� pilnai
4) ? (neveikia atomicAdd, tod�l spausdina tik pirm�j� element�)
5) Trumpiausias vienos gijos kodas buvo programoje 1b
6) Intel Core i7-3610QM 4 branduoliai 2.3GHz, OA - 12GB DDR3, OS - Microsoft Windows 10, NVIDIA - Geforce GT 635m 2gb 
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string>

using namespace std;

#define MAX_SIZE 20
#define MAX_PAV_LENGTH 30

__device__ int dev_count;

 /*Rezultat� strukt�ra
 id - id
 pav - automobilio modelis
 metai - automobilio pagaminimo metai
 v - automobilio variklio t�ris*/
struct Rez{
	int id;
	char pav[MAX_PAV_LENGTH];
	int metai;
	double v;
};

/* 
Duomen� skaitymo i� failo funkcija
@param S - automobili� modeli� pavadinim� masyvas
@param I - automobili� pagaminimo met� masyvas
@param D - automobilio variklio t�ri� masyvas
@param n - nuskaityt� duomen� kiekis
*/
void Skaityti(char *S, int *I, double *D, int *n);
/*
Rezultat� spausdinimo funkcija
@param S - automobili� modeli� pavadinim� masyvas
@param I - automobili� pagaminimo met� masyvas
@param D - automobilio variklio t�ri� masyvas
@param P - rezultat� strukt�r� masyvas
@param n - nuskaityt� duomen� kiekis
*/
void Spausdinti(char *S, int *I, double *D, Rez *P, int *n);

/*
GPU lygiagre�iai vykdoma rezultat� masyvo pildymo funkcija
@param S - automobili� modeli� pavadinim� masyvas
@param I - automobili� pagaminimo met� masyvas
@param D - automobilio variklio t�ri� masyvas
@param P - rezultat� strukt�r� masyvas
@param n - nuskaityt� duomen� kiekis
*/
__global__ void Pildyti(const char *S, const int *I, const double *D, Rez *P, const int *n);

int main()
{
	char *S;
	int *I;
	double *D;
	int *n;
	int count = 0;

	S = (char *)malloc(MAX_SIZE * MAX_PAV_LENGTH * sizeof(char*));
	I = (int *)malloc(MAX_SIZE * sizeof(int));
	D = (double *)malloc(MAX_SIZE * sizeof(double));
	n = (int *)malloc(MAX_SIZE * sizeof(int));

	(*n) = 0;

	// rezultat� masyvo inicijavimas
	Rez *P = new Rez[MAX_SIZE];
	for (int i = 0; i < MAX_SIZE; i++){
		strcpy(P[i].pav, "xxx");
		P[i].id = -1;
		P[i].metai = -1;
		P[i].v = -1.1;
	}

	Skaityti(S, I, D, n);

	// GPU kintam�j� suk�rimas
	char *dev_S;
	int *dev_I;
	double *dev_D;
	Rez *dev_P;
	int *dev_n;
	hipError_t cudaStatus;


	// Pasirenkamas GPU (jei kompiuteryje yra daugiau nei vienas)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Atminties i�skyrimas
	cudaStatus = hipMalloc((void **)&dev_S, MAX_SIZE * MAX_PAV_LENGTH * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_I, MAX_SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_D, MAX_SIZE * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_P, MAX_SIZE * sizeof(Rez));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_n, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Kopijuoja duomenis i� CPU � GPU
	cudaStatus = hipMemcpy(dev_S, S, MAX_SIZE * MAX_PAV_LENGTH * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_I, I, MAX_SIZE * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_D, D, MAX_SIZE * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P, P, MAX_SIZE * sizeof(Rez), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_count), &count, sizeof(int), 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "__global__ hipMemcpy failed!");
		goto Error;
	}

	// Funkcija vykdoma lygiagre�iai
	Pildyti << <1, (*n)>> >(dev_S, dev_I, dev_D, dev_P, dev_n);

	hipDeviceSynchronize();

	// Tikrinama d�l klaid� branduolyje
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Pildyti launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize laukia, kol branduolys baigs darb�, tikrina d�l klaid�
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Pildyti!\n", cudaStatus);
		goto Error;
	}

	// Kopijuojamami rezultatai i� GPU � CPU
	cudaStatus = hipMemcpy(P, dev_P, MAX_SIZE * sizeof(Rez), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipMemcpyFromSymbol(&count, HIP_SYMBOL(&dev_count), sizeof(int), hipMemcpyDeviceToHost);

	// Jei programos vykdymo metu atsirado klaid�, programa nukreipiama �ia
Error:
	hipFree(dev_S);
	hipFree(dev_I);
	hipFree(dev_D);
	hipFree(dev_P);
	hipFree(dev_n);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }

	Spausdinti(S, I, D, P, n);

    return 0;
}

void Skaityti(char *S, int *I, double *D, int *n){
	FILE *file = fopen("KazlauskasM_L1c_dat.txt", "r");
	char buferis[MAX_PAV_LENGTH];
	// Skaito, kol nepasiekiama failo pabaiga
	while (!feof(file)){
		int pozicija = (*n) * MAX_PAV_LENGTH;

		// Skaito i� failo iki pirmo rasto tarpo (modelis)
		fscanf(file, "%s", buferis);

		for (int i = pozicija; i < pozicija + MAX_PAV_LENGTH; i++){
			S[i] = buferis[i - pozicija];
		}

		S[pozicija + MAX_PAV_LENGTH - 1] = '\0';

		// Skaito i� failo iki pirmo rasto tarpo (metai)
		fscanf(file, "%s", buferis);
		sscanf(buferis, "%d", &I[(*n)]);

		// Skaito i� failo iki pirmo rasto tarpo (variklio t�ris)
		fscanf(file, "%s", buferis);
		sscanf(buferis, "%lf", &D[(*n)]);

		// skaitys kitoje eilut�je
		(*n)++;
	}
	fclose(file);
}

void Spausdinti(char *S, int *I, double *D, Rez *P, int *n) {
	FILE *file = fopen("KazlauskasM_L1c_rez.txt", "w");
	int iter;
	char temp[MAX_PAV_LENGTH];

	fprintf(file, "Duomen� rinkinys\n");
	fprintf(file, "-----------------------------------\n");
	fprintf(file, "    Modelis     Metai Variklio t�ris\n");
	for (int i = 0; i < (*n); i++) {
		if (i < 9) {
			fprintf(file, "0%d) ", i + 1);
		}
		else {
			fprintf(file, "%d) ", i + 1);
		}

		iter = i * MAX_PAV_LENGTH;
		for (int j = iter; j < iter + MAX_PAV_LENGTH; j++) {
			temp[j - iter] = S[j];
		}
		fprintf(file, "%-12s %-3d %6.2f\n", temp, I[i], D[i]);
	}
	fprintf(file, "-----------------------------------\n");
	fprintf(file, "Rezultatai\n");
	fprintf(file, "-----------------------------------\n");
	fprintf(file, "    Modelis     Metai Variklio t�ris\n");
	for (int i = 0; i < (*n); i++) {
		if (i < 9) {
			fprintf(file, "0%d) ", i + 1);
		}
		else {
			fprintf(file, "%d) ", i + 1);
		}
		fprintf(file, "%-12s %-3d %6.2f\n", P[i].pav, P[i].metai, P[i].v);
	}

	fclose(file);
}

__global__ void Pildyti(const char *S, const int *I, const double *D, Rez *P, const int *n){
	int id = threadIdx.x;
	if (id < 10)
		// Papildomas darbas
		for (int i = 0; i < 500000; i++){
			double x = 1000 * 20000 / i + i;
		}
	if (id < n[0]){
		int pozicija = id * 30;
		for (int i = pozicija; i < pozicija + 30; i++) P[dev_count].pav[i - pozicija] = S[i];
		P[dev_count].id = id;
		P[dev_count].metai = I[id];
		P[dev_count].v = D[id];
	}
	atomicAdd(&dev_count, 1);
}